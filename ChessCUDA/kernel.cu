﻿
// the cuda library
#include "hip/hip_runtime.h"
#include ""

// the chess library and its components
#include "chess_board.h"
#include <algorithm>
#include <unordered_map>

// the web stuff ;-;
#include <Poco/URI.h>
#include <Poco/Net/HTTPClientSession.h>
#include <Poco/Net/HTTPRequest.h>
#include <Poco/Net/HTTPResponse.h>
#include <Poco/Net/HTTPMessage.h>


std::unordered_map<std::string, std::vector<Move>> calculateAllMoves(ChessBoard board);

void sendFEN(ChessBoard board);
void sendMoves(std::unordered_map<std::string, std::vector<Move>> moves);
Move getPlayed();

int main()
{
    ChessBoard board = ChessBoard();

	sendFEN(board);

    auto allMoves = calculateAllMoves(board);

    sendMoves(allMoves);

	Move move = getPlayed();

    if(move.isValid()){
        board.makeMove(move);

		std::cout << move << std::endl;
        std::cout << board << std::endl;

        sendFEN(board);
    }

    return 0;
}

std::unordered_map<std::string, std::vector<Move>> calculateAllMoves(ChessBoard board) {

    std::unordered_map<std::string, std::vector<Move>> allMoves;

    for (int i = 0; i < 64; ++i) {
        if (board.board[i] != Piece::NONE) {
            std::string loc = std::string(1, (i % 8) + 'a') + std::to_string(i / 8 + 1);

            allMoves[loc] = board.getMoves(i);
        }
    }

    return allMoves;
}


void sendFEN(ChessBoard board) {
    std::string fen = board.toFEN();
    std::replace(fen.begin(), fen.end(), ' ', '_');

    std::string url = "https://localhost:5000";
    
    Poco::URI uri(url);
    Poco::Net::HTTPClientSession session(uri.getHost(), uri.getPort());
    
    session.setKeepAlive(true);

    Poco::Net::HTTPRequest request(Poco::Net::HTTPRequest::HTTP_GET, "/set/board/" + fen, Poco::Net::HTTPMessage::HTTP_1_1);

    session.sendRequest(request);

    Poco::Net::HTTPResponse response;
    std::istream& page = session.receiveResponse(response);

    std::cout << "SEND FEN\t- HTTP Status: " << response.getStatus() << "\n";
    
    session.setKeepAlive(false);
}

void sendMoves(std::unordered_map<std::string, std::vector<Move>> const moves) {

    std::string url = "https://localhost:5000";

    std::string parsedMoves = "";

    for (auto const& piece : moves) {
        parsedMoves += piece.first;
        for (Move move : piece.second) {
            parsedMoves += move.to_string(true);
        }
        parsedMoves += "_";
    }

    Poco::URI uri(url);
    Poco::Net::HTTPClientSession session(uri.getHost(), uri.getPort());

    session.setKeepAlive(true);

    Poco::Net::HTTPRequest request(Poco::Net::HTTPRequest::HTTP_GET, "/set/moves/" + parsedMoves, Poco::Net::HTTPMessage::HTTP_1_1);
    
    session.sendRequest(request);

    Poco::Net::HTTPResponse response;
    std::istream& page = session.receiveResponse(response);

    std::cout << "SEND MOVES\t- HTTP Status: " << response.getStatus() << "\n";

    session.setKeepAlive(false);
}

Move getPlayed() {

	std::string url = "https://localhost:5000";

	Poco::URI uri(url);
	Poco::Net::HTTPClientSession session(uri.getHost(), uri.getPort());

	session.setKeepAlive(true);

	Poco::Net::HTTPRequest request(Poco::Net::HTTPRequest::HTTP_GET, "/get/play", Poco::Net::HTTPMessage::HTTP_1_1);

	session.sendRequest(request);

	Poco::Net::HTTPResponse response;
	std::istream& page = session.receiveResponse(response);

	std::cout << "GET PLAYED\t- HTTP Status: " << response.getStatus() << "\n";

    std::string str(std::istreambuf_iterator<char>(page), {});

	session.setKeepAlive(false);

    return Move(str);
}

