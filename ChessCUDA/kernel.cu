﻿
// the cuda library
#include "hip/hip_runtime.h"
#include ""

// the chess library and its components
#include "chess_board.h"
#include <algorithm>

// the web stuff ;-;
#include <Poco/URI.h>
#include <Poco/Net/HTTPClientSession.h>
#include <Poco/Net/HTTPRequest.h>
#include <Poco/Net/HTTPResponse.h>
#include <Poco/Net/HTTPMessage.h>

void sendFEN(ChessBoard board);
void sendMoves(ChessBoard board);

int main()
{
    std::string test = "8/8/P2P2P1/P1PqP1P1/P1P1P1P1/P2P2P1/PP4PP/8 w - - 0 1";

    ChessBoard board = ChessBoard();

    std::vector<Move> moves = board.getMoves(10);

    std::cout << board.toFEN();

    std::cout << "\n{";
    for(Move m : moves) {
        std::cout << m << ", ";
    }
    std::cout << "}\n";

	sendFEN(board);

    return 0;
}

void sendFEN(ChessBoard board) {
    std::string fen = board.toFEN();
    std::replace(fen.begin(), fen.end(), ' ', '_');

    std::string url = "https://localhost:5000";
    
    Poco::URI uri(url);
    Poco::Net::HTTPClientSession session(uri.getHost(), uri.getPort());
    
    session.setKeepAlive(true);

    Poco::Net::HTTPRequest request(Poco::Net::HTTPRequest::HTTP_GET, "/board/" + fen, Poco::Net::HTTPMessage::HTTP_1_1);

    session.sendRequest(request);

    Poco::Net::HTTPResponse response;
    std::istream& page = session.receiveResponse(response);

    std::cout << "HTTP Status: " << response.getStatus() << "\n";
    
    session.setKeepAlive(false);
}

void sendMoves(ChessBoard board) {
    std::string url = "https://localhost:5000";

    Poco::URI uri(url);
    Poco::Net::HTTPClientSession session(uri.getHost(), uri.getPort());
}